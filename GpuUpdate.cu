#include "hip/hip_runtime.h"
#include "VertexInput.h"
#include <vector>

#include "GpuUpdate.h"

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void UpdateNodes(VertexData* vertices, size_t nrOfVerts, float* apPlot, size_t apPlotSize, float apMinValue, float apd, float diastolicInterval, float deltaTimeInMs, float deltaTime, float dist)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= nrOfVerts)
		return;

	VertexData& vertex = vertices[i];

	switch (vertex.state)
	{
	case State::APD:
		vertex.timePassed += deltaTimeInMs;

		int idx = int(vertex.timePassed);

		if (idx > 0 && idx < apPlotSize && (size_t(idx) + size_t(1)) < apPlotSize)
		{
			float value1 = apPlot[idx];
			float value2 = apPlot[(size_t(idx) + size_t(1))];
			float t = vertex.timePassed - idx;

			float lerpedValue = value1 + t * (value2 - value1);

			float valueRange01 = (lerpedValue - apMinValue) / dist;

			vertex.actionPotential = lerpedValue;
		}

		if (vertex.timePassed >= apd)
		{
			vertex.timePassed = 0.f;
			vertex.state = State::DI;
		}

		break;
	case State::DI:
		vertex.timePassed += deltaTimeInMs;

		if (vertex.timePassed >= diastolicInterval)
		{
			vertex.timePassed = 0.f;
			vertex.state = State::Waiting;
		}
		break;
	}

}


CudaUpdate::~CudaUpdate()
{
	if (m_DeviceVerts != nullptr)
		hipFree(m_DeviceVerts);
	if (m_DeviceApPlot != nullptr)
		hipFree(m_DeviceApPlot);

}

void CudaUpdate::Update(std::vector<VertexData>& vertices, std::vector<float>& apPlot, float apMinValue, float apd, float diastolicInterval, float deltaTimeInMs, float deltaTime, float dist)
{
	hipError_t err = hipSuccess;
	if (m_DeviceVerts == nullptr)
	{
		err = hipMalloc((void**)&m_DeviceVerts, vertices.size() * sizeof(VertexData));
	}
	hipMemcpy(m_DeviceVerts, vertices.data(), vertices.size() * sizeof(VertexData), hipMemcpyHostToDevice);
	if (m_DeviceApPlot == nullptr)
	{
		err = hipMalloc((void**)&m_DeviceApPlot, apPlot.size() * sizeof(float));
	}
	hipMemcpy(m_DeviceApPlot, apPlot.data(), apPlot.size() * sizeof(float), hipMemcpyHostToDevice);


	int threadsPerBlock{256};
	int numBlocks{ (int(vertices.size()) + threadsPerBlock - 1) / threadsPerBlock };


	UpdateNodes <<<numBlocks, threadsPerBlock>>>(m_DeviceVerts, vertices.size(), m_DeviceApPlot, apPlot.size(), apMinValue, apd, diastolicInterval, deltaTimeInMs, deltaTime, dist);

	hipMemcpy(vertices.data(), m_DeviceVerts, vertices.size() * sizeof(VertexData), hipMemcpyDeviceToHost);

}
