#include "hip/hip_runtime.h"
#include "VertexInput.h"
#include <vector>

#include "GpuUpdate.h"

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void UpdateNodes(VertexInput* vertices, size_t nrOfVerts, float* apPlot, size_t apPlotSize, float apMinValue, float apd, float diastolicInterval, float deltaTimeInMs, float deltaTime, float dist)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= nrOfVerts)
		return;

	VertexInput& vertex = vertices[i];

	switch (vertex.state)
	{
	case VertexInput::State::APD:
		vertex.timePassed += deltaTimeInMs;

		int idx = int(vertex.timePassed);

		if (idx > 0 && idx < apPlotSize && (size_t(idx) + size_t(1)) < apPlotSize)
		{
			float value1 = apPlot[idx];
			float value2 = apPlot[(size_t(idx) + size_t(1))];
			float t = vertex.timePassed - idx;

			float lerpedValue = value1 + t * (value2 - value1);

			float valueRange01 = (lerpedValue - apMinValue) / dist;

			vertex.actionPotential = lerpedValue;
			vertex.apVisualization = valueRange01;
		}

		if (vertex.timePassed >= apd)
		{
			vertex.timePassed = 0.f;
			vertex.state = VertexInput::State::DI;
			vertex.apVisualization = 0.f;
		}

		break;
	case VertexInput::State::DI:
		vertex.timePassed += deltaTimeInMs;

		if (vertex.timePassed >= diastolicInterval)
		{
			vertex.timePassed = 0.f;
			vertex.state = VertexInput::State::Waiting;
		}
		break;
	}

}


CudaUpdate::~CudaUpdate()
{
	if (m_DeviceVerts != nullptr)
		hipFree(m_DeviceVerts);
	if (m_DeviceApPlot != nullptr)
		hipFree(m_DeviceApPlot);

}

void CudaUpdate::Update(std::vector<VertexInput>& vertices, std::vector<float>& apPlot, float apMinValue, float apd, float diastolicInterval, float deltaTimeInMs, float deltaTime, float dist)
{
	hipError_t err = hipSuccess;
	if (m_DeviceVerts == nullptr)
	{
		err = hipMalloc((void**)&m_DeviceVerts, vertices.size() * sizeof(VertexInput));
	}
	hipMemcpy(m_DeviceVerts, vertices.data(), vertices.size() * sizeof(VertexInput), hipMemcpyHostToDevice);
	if (m_DeviceApPlot == nullptr)
	{
		err = hipMalloc((void**)&m_DeviceApPlot, apPlot.size() * sizeof(float));
	}
	hipMemcpy(m_DeviceApPlot, apPlot.data(), apPlot.size() * sizeof(float), hipMemcpyHostToDevice);


	int threadsPerBlock{256};
	int numBlocks{ (int(vertices.size()) + threadsPerBlock - 1) / threadsPerBlock };


	UpdateNodes <<<numBlocks, threadsPerBlock>>>(m_DeviceVerts, vertices.size(), m_DeviceApPlot, apPlot.size(), apMinValue, apd, diastolicInterval, deltaTimeInMs, deltaTime, dist);

	hipMemcpy(vertices.data(), m_DeviceVerts, vertices.size() * sizeof(VertexInput), hipMemcpyDeviceToHost);

}
